#include "hip/hip_runtime.h"
#include"mykernels.h"
#include"pngwriter.h"
#include <iostream>
#include<string>
#include<algorithm> 
#include<cmath>
#include<sstream>
#include<iomanip>
int dirArr2[]={0,0,0,-1,0,1,-1,0,-1,-1,-1,1,1,0,1,-1,1,1};
const int fsize = sizeof(float);
using namespace std;
float * feq, *fin, *feq0,*rho,*momentum,*cresult, * fout,*result;
void preparations(){
  allocConstants();
  hipMalloc(&feq,_N*_Q*fsize);
  hipMalloc(&feq0,_N*_Q*fsize);
  hipMalloc(&fin,_N*_Q*fsize);
  result = new float[_N];
  hipMalloc(&fout,_N*_Q*fsize);
  hipMalloc(&rho,_N*fsize);
   hipMalloc(&momentum,_N*2*fsize);
   hipMalloc(&cresult,_N*fsize);
  equilibriumInit<<<_Ny,_Nx>>>(feq);
  hipMemcpy(feq0,feq,_N*_Q*fsize,hipMemcpyDeviceToDevice);
  hipMemcpy(fin,feq,_N*_Q*fsize,hipMemcpyDeviceToDevice);
  
  
}
void cleaning(){
  hipFree(feq);
  hipFree(fout);
  hipFree(feq0);
  hipFree(fin);
  hipFree(rho);
  hipFree(momentum);
  delete [] result;
  hipFree(cresult);
}

string itoswithzeros(int i1, int len){
stringstream ss;
ss << setw(len) << setfill('0') << i1;
return ss.str();
}


void step(){
  

  findRhoAndMomentum<<<_Ny,_Nx>>>(fin,rho, momentum);
  prepareResult<<<_Ny,_Nx>>>(momentum,cresult);
  hipMemcpy(result, cresult, _N*fsize, hipMemcpyDeviceToHost);
  equilibrium<<<_Ny,_Nx>>>(rho,momentum, feq);
  findFout<<<_Ny,_Nx>>>(fin, feq, fout);
  for(int a =0;a<9;++a){
    npRoll<<<_Ny,_Nx>>>(fout+a*_N,fin+a*_N,_Nx,_Ny,-dirArr2[2*a],-dirArr2[2*a+1]);
   copyFirstRow<<<1,_Ny>>>(fin, feq0);
  }
  applyObstacle<<<_Ny,_Nx>>>(fin,fout);
  
}
void printArray(float* arr){
  cout<<"\n";
  for(int a=0;a< _Ny;++a){
    cout<<"\n";
    for(int b =0;b< _Nx;++b)
      cout<<" "<<arr[a*_Nx+b];
  }
}
int main(){
  string s = "images/img";
 preparations();
 for(int a =0;a<700000;++a) {
   step();
   if(a %200 == 0){
    cout<<a<<" ";
     prepareResult<<<_Ny,_Nx>>>(momentum,cresult);
     hipMemcpy(result, cresult, _N*fsize, hipMemcpyDeviceToHost);
   // printArray(result);
     pngwriter out(_Nx, _Ny, 0, (s+itoswithzeros(a/200,4)+".png").c_str());
     float max1=0;
     for(int b =0 ;b< _N;++b) if(max1 < abs(result[b])) max1 = abs(result[b]);
     for(int i = 0; i<_Ny; ++i)
       for(int j = 0;j< _Nx;++j)
	 out.plot(j, i, 0.0, 1.0-result[i*_Nx+j] /(max1),1.0-result[i*_Nx+j] /(max1));
     out.close();
  }
 }
 
 cleaning();
}

void drawPng(string path, float * result){
  
}
