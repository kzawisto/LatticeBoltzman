#include "hip/hip_runtime.h"
#include<mykernels.h>
#include <cstdio>
#include<cmath>
int dirArr[]={0,0,0,-1,0,1,-1,0,-1,-1,-1,1,1,0,1,-1,1,1};
int revArr[]={0, 2, 1, 6, 8, 7, 3, 5, 4};
float wArr[]={4.f/9.f,1.f/9.f,1.f/9.f,1.f/9.f,1.f/36.f,1.f/36.f,1.f/9.f,1.f/36.f,1.f/36.f};
__constant__ int cdirArr [18], crevArr[9];
__constant__ float cwArr[9];

__global__ void npRoll(float * arr1, float * arr2, int w, int h,int stepx, int stepy){
  int i =blockIdx.x;
   arr2[i*w+threadIdx.x] = arr1[((i+stepy+h)%h)*w + (threadIdx.x+stepx+w)%w];
  
}



__global__ void equilibrium(float * rho, float * vel,float * feq){
  const int b = blockIdx.x, t = threadIdx.x;
  float v;
  float usq =vel[b *_Nx + t] * vel[b *_Nx + t] + vel[_N+ b *_Nx + t]*vel[_N+ b *_Nx + t];
  #pragma unroll
  for(int i =0;i< 9;++i){
    v=cdirArr[2*i] *vel[b *_Nx + t] + cdirArr[2*i+1] *vel[_N+ b *_Nx + t];
    feq[i*_N+ b *_Nx + t]= rho[b *_Nx + t] * cwArr[i] *(1.f + 3* v +4.5f*v*v- 1.5f*usq);
  }
}
__global__ void equilibriumInit(float * feq){
  const int b = blockIdx.x, t = threadIdx.x;
  float v, vX =  _ULB *(1.0+1e-2f*sin((3*(float)blockIdx.x)/(_Ny-1)*M_PI));
 #pragma unroll
  for(int i =0;i< 9;++i){
    v=cdirArr[2*i] *vX;
   
    feq[i*_N+ b *_Nx + t]= cwArr[i] *(1.f + 3* v +4.5f*v*v- 1.5f*vX*vX);
  }
}


__global__ void findRhoAndMomentum(float* fin, float * rho, float* momentum){
    const int b = blockIdx.x, t = threadIdx.x;
    float r=0, m1=0,m2=0;
    #pragma unroll
    for(int i =0; i< 9;++i){
      r+=fin[i*_N+b*_Nx+t];
      m1 += fin[i*_N+b*_Nx+t]*cdirArr[2*i];
      m2 += fin[i*_N+b*_Nx+t]*cdirArr[2*i+1];
    }
    momentum[_N+b*_Nx+t] = m2/r;
    momentum[b*_Nx+t] = m1/r;
    rho[b*_Nx+t] = r;
    //rho[b*blockDim.x +t]=
  
}
__global__ void findFout(float* fin, float * feq, float* fout){
     const int b = blockIdx.x, t = threadIdx.x;
     #pragma unroll
    for(int i =0;i< 9;++i){
     fout[i * _N+b*  _Nx+t] = fin[i * _N+b * _Nx+t] - (fin[i*_N+b*_Nx+t] -feq[i*_N+b*_Nx+t])/ _TAU;
    }
}

__global__ void writeZeros(float* arr){
  const int b = blockIdx.x, t = threadIdx.x;
  arr[b*blockDim.x + t] =0;
}

__global__ void applyObstacle(float * fin, float *fout){
  const int i =blockIdx.x, j = threadIdx.x;
     if(( (i-_Cy)*(i-_Cy) + (j-_Cx)*(j-_Cx) < _RAD* _RAD)){
       #pragma unroll
       for(int k =0; k <9 ;++k){
	 fin[k*_N + i * _Nx + j]=fout[crevArr[k]*_N + i * _Nx + j];
       }
     }
}
__global__ void findResult(float * momentum, float * result){
 // const int i =blockIdx.x, j = threadIdx.x;
  
}
__global__ void prepareResult(float* momentum, float*cresult){
   const int b =blockIdx.x, t = threadIdx.x;
   cresult[b*_Nx+t]  = sqrt(momentum[_N+b*_Nx+t] * momentum[_N+b*_Nx+t] +momentum[b*_Nx+t] * momentum[b*_Nx+t]); 
}
__global__ void copyFirstRow(float *fin, float * feq){
  #pragma unroll
    for(int a =0 ;a <9;++a){
      fin[a*_N + threadIdx.x*_Nx] =feq[a*_N + threadIdx.x*_Nx] ;
    }
}
void allocConstants(){
   hipMemcpyToSymbol(HIP_SYMBOL(cdirArr),dirArr, 18*4 );  
   hipMemcpyToSymbol(HIP_SYMBOL(crevArr),revArr, 9*4 );
   hipMemcpyToSymbol(HIP_SYMBOL(cwArr),wArr, 9*4 );
}